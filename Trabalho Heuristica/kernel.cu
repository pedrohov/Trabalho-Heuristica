
#include "hip/hip_runtime.h"
#include ""
#include "calc_distancia.h"
#include <stdio.h>
#include "Simulated_Annealing.h"

int main()
{
	node *lista;
	lista = (node *)malloc(sizeof(node));

	ponto *pontos_alcance;
	pontos_alcance = (ponto *)malloc(sizeof(ponto));

	if (!lista) {
		printf("Sem memoria disponivel!\n");
		exit(1);
	}

	if (!pontos_alcance) {
		printf("Sem memoria disponivel!\n");
		exit(1);
	}

	inicia(lista);
	printf("iniciada listas de lista\n");
	//system("pause");

	inicia_pontos(pontos_alcance);
	printf("iniciada listas pontos\n");
	//system("pause");

	FILE *arq_pontos;
	FILE *arq_demanda;
	
	int x, y, demanda = 0;
	int quantidade_pontos = 0;
	int raio = 0;
	int facilidades = 0;
	
	arq_pontos = fopen("Dataset/SJC324.dat", "r");
	arq_demanda = fopen("Dataset/demand-SJC324.dat", "r");

	if ((arq_pontos == NULL) ||(arq_demanda == NULL)) {

		printf("Erro, nao foi possivel abrir o arquivo\n");
	}
	else {

		fscanf(arq_pontos, "%d %d %d\n", &quantidade_pontos, &facilidades, &raio);
		
		printf("quant. pontos: %d, quant. facilidades: %d, raio: %d\n", quantidade_pontos, facilidades, raio);
		
		for (int i = 0; i < quantidade_pontos; i++) {
			fscanf(arq_demanda, "%d\n", &demanda);
			fscanf(arq_pontos, "%d %d\n", &x, &y);
			insereFim(lista, x, y, demanda);
		}
	}

	//exibe(lista);

	printf("lista de pontos preenchida\n");
	//system("pause");

	for (int i = 0; i < quantidade_pontos; i++) {
		for (int j = 0; j < quantidade_pontos; j++) {
			
			if (calcula_distancia( numPos(lista, i).cord_x, numPos(lista, i).cord_y, numPos(lista, j).cord_x, numPos(lista, j).cord_y, raio)) {
				
				//printf("dentro do raio");
				//system("pause");
			
				insereFim_pontos(numPos(lista, i).raio, numPos(lista, j).cord_x, numPos(lista, j).cord_y);
			}
		}
	}
	

	exibe(lista);

	/*SA*/
	simulated_annealing(lista, facilidades);

	/*GRASP*/

	fclose(arq_demanda);
	fclose(arq_pontos);

	free(lista);
	free(pontos_alcance);

	system("pause");
}
